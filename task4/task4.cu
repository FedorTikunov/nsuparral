#include "hip/hip_runtime.h"
﻿
#include ""
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <ctime>
#include <string>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#define CORN1 10.0
#define CORN2 20.0
#define CORN3 30.0
#define CORN4 20.0


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}




__global__ void calculationMatrix(double* new_arry, const double* old_array)
{

    int blockIndex = blockIdx.x + gridDim.y * blockIdx.y;
    int threadIndex = threadIdx.x + threadIdx.y * blockDim.x;


    int arrayIndex = blockIndex * blockDim.x * blockDim.y + threadIndex;
    int  GRID_SIZEX = gridDim.x * blockDim.x;
    int  GRID_SIZEY = gridDim.y * blockDim.y;
    int i = arrayIndex / GRID_SIZEX;
    int j = arrayIndex % GRID_SIZEX;
    if (i != 0 && i != GRID_SIZEY - 1 && j != 0 && j != GRID_SIZEX - 1) {
        new_arry[i * GRID_SIZEX + j] = 0.25 * (old_array[(i + 1) * GRID_SIZEX + j] + old_array[(i - 1) * GRID_SIZEX + j] + old_array[i * GRID_SIZEX + j - 1] + old_array[i * GRID_SIZEX + j + 1]);
        //printf("(%lf - %lf)\n", old_array[i * GRID_SIZEX + j], new_arry[i * GRID_SIZEX + j]);
    }
}

__global__ void getDifferenceMatrix(const double* new_arry, const double* old_array, double* dif)
{
    int blockIndex = blockIdx.x + gridDim.y * blockIdx.y;
    int threadIndex = threadIdx.x + threadIdx.y * blockDim.x;


    int arrayIndex = blockIndex * blockDim.x * blockDim.y + threadIndex;
    int  GRID_SIZEX = gridDim.x * blockDim.x;
    int  GRID_SIZEY = gridDim.y * blockDim.y;
    int i = arrayIndex / GRID_SIZEX;
    int j = arrayIndex % GRID_SIZEX;
    if (i != 0 && i != GRID_SIZEY - 1 && j != 0 && j != GRID_SIZEX - 1) {
        //printf("%lf = abs(%lf - %lf)\n", dif[i * GRID_SIZEX + j], old_array[i * GRID_SIZEX + j], new_arry[i * GRID_SIZEX + j]);
        dif[i * GRID_SIZEX + j] = std::abs(old_array[i * GRID_SIZEX + j] - new_arry[i * GRID_SIZEX + j]);
    }
}
__global__ void WeakcalculationMatrix(double* new_arry, const double* old_array, int size)
{

    int  i = blockIdx.x;
    int  j = threadIdx.x;

    if (i != 0 && i != size -1 && j != 0 && j != size - 1) {
        new_arry[i * size + j] = 0.25 * (old_array[(i + 1) * size + j] + old_array[(i - 1) * size + j] + old_array[i * size + j - 1] + old_array[i * size + j + 1]);

    }
}

int main(int argc, char** argv) {
    int GRID_SIZE = std::stoi(argv[2]);
    double ACC = std::pow(10, -(std::stoi(argv[1])));
    int ITER = std::stoi(argv[3]);
    double* newa = new double[GRID_SIZE * GRID_SIZE];
    double* olda = new double[GRID_SIZE * GRID_SIZE];


    std::memset(olda, 0, GRID_SIZE * GRID_SIZE * sizeof(double));

    int iter_count = 0;
    double error = 1.0;

    double prop1 = (CORN2 - CORN1) / (GRID_SIZE);
    double prop2 = (CORN3 - CORN1) / (GRID_SIZE);
    double prop3 = (CORN4 - CORN3) / (GRID_SIZE);
    double prop4 = (CORN2 - CORN4) / (GRID_SIZE);

    olda[0] = CORN1;
    olda[(GRID_SIZE - 1) * GRID_SIZE] = CORN3;
    olda[GRID_SIZE - 1] = CORN2;
    olda[GRID_SIZE - 1 + GRID_SIZE * (GRID_SIZE - 1)] = CORN4;
    newa[0] = CORN1;
    newa[(GRID_SIZE - 1) * GRID_SIZE] = CORN3;
    newa[GRID_SIZE - 1] = CORN2;
    newa[GRID_SIZE - 1 + GRID_SIZE * (GRID_SIZE - 1)] = CORN4;

    double* d_newa,* d_olda, *d_dif;
    hipMalloc((void**)&d_olda, sizeof(double) * GRID_SIZE * GRID_SIZE);
    hipMalloc((void**)&d_newa, sizeof(double) * GRID_SIZE * GRID_SIZE);
    hipMalloc((void**)&d_dif, sizeof(double) * GRID_SIZE * GRID_SIZE);


    clock_t beforeinit = clock();
    for (size_t i = 1; i < GRID_SIZE - 1; i++) {
        olda[i] = olda[0] + prop1 * i;
        olda[i * GRID_SIZE] = olda[0] + prop2 * i;
        olda[(GRID_SIZE - 1) * GRID_SIZE + i] = olda[(GRID_SIZE - 1) * GRID_SIZE] + prop3 * i;
        olda[GRID_SIZE * i + GRID_SIZE - 1] = olda[GRID_SIZE * (GRID_SIZE - 1) + GRID_SIZE - 1] + prop4 * i;
        newa[i] = olda[i];
        newa[i * GRID_SIZE] = olda[i * GRID_SIZE];
        newa[(GRID_SIZE - 1) * GRID_SIZE + i] = olda[(GRID_SIZE - 1) * GRID_SIZE + i];
        newa[GRID_SIZE * i + GRID_SIZE - 1] = olda[GRID_SIZE * i + GRID_SIZE - 1];
    }

    dim3 block_dim(16, 16);
    dim3 grid_dim(GRID_SIZE / block_dim.x, GRID_SIZE/ block_dim.y);

    hipMemcpy(d_olda, olda, sizeof(double) * GRID_SIZE * GRID_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_newa, newa, sizeof(double) * GRID_SIZE * GRID_SIZE, hipMemcpyHostToDevice);
    double* max_error = 0;
    hipMalloc((void**)&max_error, sizeof(double));
    std::cout << "Initialization time: " << 1.0 * (clock() - beforeinit) / CLOCKS_PER_SEC << std::endl;
    size_t temp_storage_bytes = 0;
    double* temp_storage = NULL;
    hipcub::DeviceReduce::Max(temp_storage, temp_storage_bytes, d_dif, max_error, GRID_SIZE * GRID_SIZE);
    hipMalloc((void**)&temp_storage, temp_storage_bytes);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    double beta = -1.0;
    while (iter_count < ITER && error > ACC) {
        iter_count++;
        calculationMatrix <<<grid_dim, block_dim >>> (d_newa, d_olda);

        if(iter_count % 100 == 0){
            getDifferenceMatrix <<<grid_dim, block_dim >>> (d_newa, d_olda, d_dif);
            hipcub::DeviceReduce::Max(temp_storage, temp_storage_bytes, d_dif, max_error, GRID_SIZE * GRID_SIZE);
            hipMemcpy(&error, max_error, sizeof(double), hipMemcpyDeviceToHost);
            error = std::abs(error);
        }

        double* c = d_olda;
        d_olda = d_newa;
        d_newa = c;
    }
    
    /*
    hipMemcpy((void*)olda, (void*)d_olda, sizeof(double) * GRID_SIZE * GRID_SIZE, hipMemcpyDeviceToHost);

    for (size_t i = 0; i < GRID_SIZE; i++)
    {
        for (size_t j = 0; j < GRID_SIZE; j++)
        {
            std::cout << olda[i * GRID_SIZE + j] << " ";
        }
        std::cout << std::endl;
    }
    */
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);
    std::cout << "Calculation time: " << time << std::endl;
    std::cout << "Iteration: " << iter_count << " " << "Error: " << error << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_olda);
    hipFree(d_newa);
    hipFree(temp_storage);
    delete[] olda;
    delete[] newa;
return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
