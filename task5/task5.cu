#include "hip/hip_runtime.h"
﻿
#include ""
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <ctime>
#include <string>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <mpi.h>


//значения в углах сетки
#define CORN1 10.0
#define CORN2 20.0
#define CORN3 30.0
#define CORN4 20.0

#define BlOCK_SIZE 16

//функция по подсчету/обновлению ячейк сетке
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//функция получает указатели двух массив. 
//обновляет ячейки первого массива на основе среднего значения четерех ближайших по индексу ячейк из второго массива
//функция являеться global и распоточивает подсчет матрицы на потоки
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void calculationMatrix(double* new_arry, const double* old_array, size_t size)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
    //printf("%d", size);
    if (i != 0 && i != size - 1 && j != 0 && j != size - 1)
    {
        new_arry[i * size + j] = 0.25 * (old_array[i * size + j - 1] + old_array[(i - 1) * size + j] +
            old_array[(i + 1) * size + j] + old_array[i * size + j + 1]);
    }
}
//функция по вычислению разницы матриц
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//функция получает указатели трех массив. 
//модуль разницы двух первых массивов записывает в третий
//при распоточивание, 1d массивы разбеваются на блоки по 32x32 как 2d массивы
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void getDifferenceMatrix(const double* new_arry, const double* old_array, double* dif)
{   
    int blockIndex = blockIdx.x + gridDim.y * blockIdx.y;
    int threadIndex = threadIdx.x + threadIdx.y * blockDim.x;


    int arrayIndex = blockIndex * blockDim.x * blockDim.y + threadIndex;
    int  GRID_SIZEX = gridDim.x * blockDim.x;
    int  GRID_SIZEY = gridDim.y * blockDim.y;
    int i = arrayIndex / GRID_SIZEX;
    int j = arrayIndex % GRID_SIZEX;
    if (i != 0 && i != GRID_SIZEY - 1 && j != 0 && j != GRID_SIZEX - 1) {
        //printf("%lf = abs(%lf - %lf)\n", dif[i * GRID_SIZEX + j], old_array[i * GRID_SIZEX + j], new_arry[i * GRID_SIZEX + j]);
        dif[i * GRID_SIZEX + j] = std::abs(old_array[i * GRID_SIZEX + j] - new_arry[i * GRID_SIZEX + j]);
    }
}

//основной код
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//получает из коммандной строки значения для размерность сетки, точности обновления сетки, максимального количества итераций
//выделяет память на host и device для сеток
//заполняем сетки начальными значениями
//производим вычисления на GPU
//выводим скорость вычисления, кол. итераций и точнось в консоль
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) {

    MPI_Init(&argc, &argv);
    int rank, size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    // Получаем значения из коммандной строки
    int GRID_SIZE = std::stoi(argv[2]); // размерность сетки
    double ACC = std::pow(10, -(std::stoi(argv[1]))); // до какой точность обновлять сетку
    int ITER = std::stoi(argv[3]); //  максимальное количество итераций

    //выделяем память под 2 сетки размера GRID_SIZExGRID_SIZE
    double* newa = new double[GRID_SIZE * GRID_SIZE]; 
    double* olda = new double[GRID_SIZE * GRID_SIZE];


    std::memset(olda, 0, GRID_SIZE * GRID_SIZE * sizeof(double));

    int iter_count = 0; // счетчик итераций
    double error = 1.0; // переменная ошибки
    
    double prop1 = (CORN2 - CORN1) / (GRID_SIZE);
    double prop2 = (CORN3 - CORN1) / (GRID_SIZE);
    double prop3 = (CORN4 - CORN3) / (GRID_SIZE);
    double prop4 = (CORN2 - CORN4) / (GRID_SIZE);

    //записываем значения в углы сеток
    olda[0] = CORN1;
    olda[(GRID_SIZE - 1) * GRID_SIZE] = CORN3;
    olda[GRID_SIZE - 1] = CORN2;
    olda[GRID_SIZE - 1 + GRID_SIZE * (GRID_SIZE - 1)] = CORN4;
    newa[0] = CORN1;
    newa[(GRID_SIZE - 1) * GRID_SIZE] = CORN3;
    newa[GRID_SIZE - 1] = CORN2;
    newa[GRID_SIZE - 1 + GRID_SIZE * (GRID_SIZE - 1)] = CORN4;

    //выделяем память на gpu через cuda для 3 сеток
    double* d_newa,* d_olda, *d_dif;
    hipMalloc((void**)&d_olda, sizeof(double) * GRID_SIZE * GRID_SIZE);
    hipMalloc((void**)&d_newa, sizeof(double) * GRID_SIZE * GRID_SIZE);
    hipMalloc((void**)&d_dif, sizeof(double) * GRID_SIZE * GRID_SIZE);

    //вычисления значений границ сетки
    clock_t beforeinit = clock();
    for (size_t i = 1; i < GRID_SIZE - 1; i++) {
        olda[i] = olda[0] + prop1 * i;
        olda[i * GRID_SIZE] = olda[0] + prop2 * i;
        olda[(GRID_SIZE - 1) * GRID_SIZE + i] = olda[(GRID_SIZE - 1) * GRID_SIZE] + prop3 * i;
        olda[GRID_SIZE * i + GRID_SIZE - 1] = olda[GRID_SIZE * (GRID_SIZE - 1) + GRID_SIZE - 1] + prop4 * i;
        newa[i] = olda[i];
        newa[i * GRID_SIZE] = olda[i * GRID_SIZE];
        newa[(GRID_SIZE - 1) * GRID_SIZE + i] = olda[(GRID_SIZE - 1) * GRID_SIZE + i];
        newa[GRID_SIZE * i + GRID_SIZE - 1] = olda[GRID_SIZE * i + GRID_SIZE - 1];
    }
    
    // размерность блоков и грида 
    dim3 block_dim(32, 32);
    dim3 grid_dim(GRID_SIZE / block_dim.x, GRID_SIZE/ block_dim.y);

    // Define CUDA streams for overlapping computation and communication
    hipStream_t compute_stream, comm_stream;
    hipStreamCreate(&compute_stream);
    hipStreamCreate(&comm_stream);

    // Define CUDA grid and block sizes

    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((GRID_SIZE - 2 + BLOCK_SIZE - 1) / BLOCK_SIZE, (GRID_SIZE - 2 + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Define MPI data types for exchanging boundary conditions
    MPI_Datatype col_type;
    MPI_Type_vector(GRID_SIZE - 2, 1, N, MPI_DOUBLE, &col_type);
    MPI_Type_commit(&col_type);

    MPI_Datatype row_type;
    MPI_Type_contiguous(GRID_SIZE - 2, MPI_DOUBLE, &row_type);
    MPI_Type_commit(&row_type);


    // копирование информации с CPU на GPU
    hipMemcpy(d_olda, olda, sizeof(double) * GRID_SIZE * GRID_SIZE, hipMemcpyHostToDevice); // (CPU) olda -> (GPU) d_olda
    hipMemcpy(d_newa, newa, sizeof(double) * GRID_SIZE * GRID_SIZE, hipMemcpyHostToDevice); // (CPU) newa -> (GPU) d_newa

    //выделяем память на gpu для переменной, которая будет хранить ошибку на device
    double* max_error = 0;
    hipMalloc((void**)&max_error, sizeof(double));

    std::cout << "Initialization time: " << 1.0 * (clock() - beforeinit) / CLOCKS_PER_SEC << std::endl;

    size_t temp_storage_bytes = 0;
    double* temp_storage = NULL;
    //получаем размер временного буфера для редукции
    hipcub::DeviceReduce::Max(temp_storage, temp_storage_bytes, d_dif, max_error, GRID_SIZE * GRID_SIZE);

    //выделяем память для буфера
    hipMalloc((void**)&temp_storage, temp_storage_bytes);

    clock_t beforecal = clock();
    
    //алгоритм обновления сетки, работающий пока макс. ошибка не станет меньше или равне нужной точности, или пока количество итерации не превысит максимальное количество.
    while (iter_count < ITER && error > ACC) {
        iter_count++;
        calculationMatrix << <grid, block, 0, compute_stream>>> (d_newa, d_olda, GRID_SIZE); // расчет матрицы

        // Wait for the computation to finish
        hipStreamSynchronize(compute_stream);

        if (rank % 2 == 0) {
            MPI_Sendrecv(&d_newa[1 + GRID_SIZE * (GRID_SIZE - 2)], 1, col_type, rank + 1, 0,
                &d_newa[1 + GRID_SIZE * (GRID_SIZE - 1)], 1, col_type, rank + 1, 0,
                MPI_COMM_WORLD, MPI_STATUS_IGNORE);
            MPI_Sendrecv(&d_newa[1], 1, col_type, rank - 1, 0,
                &d_newa[0], 1, col_type, rank - 1, 0,
                MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        }
        else {
            MPI_Sendrecv(&d_newa[1], 1, col_type, rank - 1, 0,
                &d_newa[0], 1, col_type, rank - 1, 0,
                MPI_COMM_WORLD, MPI_STATUS_IGNORE);
            MPI_Sendrecv(&d_newa[1 + GRID_SIZE], 1, col_type, rank + 1, 0,
                &d_newa[1 + GRID_SIZE * (GRID_SIZE - 1)], 1, col_type, rank + 1, 0,
                MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        }
        if (rank < size - 1) {
            MPI_Sendrecv(&d_newa[GRID_SIZE * (GRID_SIZE - 2) + 1], GRID_SIZE - 2, MPI_FLOAT, rank + 1, 0,
                &d_newa[GRID_SIZE * (GRID_SIZE - 1) + 1], GRID_SIZE - 2, MPI_FLOAT, rank + 1, 0,
                MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        }
        if (rank > 0) {
            MPI_Sendrecv(&d_newa[GRID_SIZE + 1], GRID_SIZE - 2, MPI_FLOAT, rank - 1, 0,
                &d_newa[1], N - 2, MPI_FLOAT, rank - 1, 0,
                MPI_COMM_WORLD, MPI_STATUS_IGNORE);
        }


        // расчитываем ошибку каждую сотую итерацию
        if(iter_count % 100 == 0){
            getDifferenceMatrix <<<grid_dim, block_dim >>> (d_newa, d_olda, d_dif); // вычисления разницы матрицы
            hipcub::DeviceReduce::Max(temp_storage, temp_storage_bytes, d_dif, max_error, GRID_SIZE * GRID_SIZE); // нахождение максимума в разнице матрицы
            hipMemcpy(&error, max_error, sizeof(double), hipMemcpyDeviceToHost); // запись ошибки в переменную на host

            error = std::abs(error);
        }

        //смена указателей между сетками на device
        double* c = d_olda;
        d_olda = d_newa;
        d_newa = c;
    }
    
    //вывод времени работы на алгоритма
    std::cout << "Calculation time: " << 1.0 * (clock() - beforecal) / CLOCKS_PER_SEC << std::endl;
    //вывод кол. итерацций и значение ошибки
    std::cout << "Iteration: " << iter_count << " " << "Error: " << error << std::endl;

    //очитска памяти
    hipFree(d_olda);
    hipFree(d_newa);
    hipFree(temp_storage);
    hipFree(max_error);
    MPI_Type_free(&col_type);
    MPI_Type_free(&row_type);
    MPI_Finalize();

    delete[] olda;
    delete[] newa;
return 0;
}

